#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <sys/mman.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <magick/MagickCore.h>
#include <hip/hip_runtime.h>
#include "upscaler.h"

int main(int argc, char** argv){
    //ensure the proper number of arguments are given
    if(argc < 2 || argc > 4){
        printf("Usage: %s <path to image> <target resolution> [path to destination]\n", argv[0]);
        exit(EXIT_FAILURE);
    }
    char* outputPath;
    if(argc == 4){
        outputPath = argv[3];
    }
    else{
        //make the default output path the current directory
        outputPath = (char*) malloc(sizeof(char)* 2);
        outputPath[0] = '.';
        outputPath[1] = '\0';
    }
    int targetResolution;
    targetResolution = atoi(argv[2]);
    
    
    //Initialize the image info structure and read an image.
    
    ExceptionInfo* exception;
    Image* image;
    Image* images;
    ImageInfo* image_info;

    //read in image from file
    MagickCoreGenesis(*argv,MagickTrue);
    exception=AcquireExceptionInfo();
    image_info=CloneImageInfo((ImageInfo *) NULL);
    (void) strcpy(image_info->filename,argv[1]);
    images=ReadImage(image_info,exception);
    if (exception->severity != UndefinedException){
        CatchException(exception);
    }
    if (images == (Image *) NULL){
        exit(1);
    }

    image = RemoveFirstImageFromList(&images);
    if(image == NULL){
        perror("No image");
        exit(EXIT_FAILURE);
    }

    //upscales the image and returns the upscaled version
    Image* upscaled_image = upscale_image(image, targetResolution, exception);

    Image* saving_list = NewImageList();
    if (upscaled_image == (Image *) NULL){
        MagickError(exception->severity,exception->reason,exception->description);
    }
    (void) AppendImageToList(&saving_list, upscaled_image);

    //Free the memory holding the input image
    DestroyImage(image);

    //write the upscaled image to disk
    (void) strcpy(saving_list->filename, outputPath);
    WriteImage(image_info, upscaled_image);

    //clean up memory
    saving_list=DestroyImageList(saving_list);
    image_info=DestroyImageInfo(image_info);
    exception=DestroyExceptionInfo(exception);
    MagickCoreTerminus();
}

/*
Procedure:           upscale_image
Parameters:          Image* image - pointer to the image to be upscaled
                     int target_resolution - the vertical resolution to which the image should be upscaled
                     ExceptionInfo* exception - ImageCore's way of tracking what might have caused an error
Purpose:             To upscale image to the target vertical resolution while maintaining the aspect ratio.
Produces:            A pointer to an image that has been upscaled.
Preconditions:       image must be loaded into memory via MagickCoreGenesis
                     target resolution must be greater than the vertical resolution of image
Postconditions:      the vertical resolution of the returned image is the smallest value greater than the
                     original resolution that is a power of two times the original resolution.
*/
Image* upscale_image(Image* image, int target_resolution, ExceptionInfo* exception) {
    Image* new_image = image;

    Image* upscaled_image;
    //make a background that can be passed to the NewMagickImage function
    MagickPixelPacket background;
    background.red = 0;
    background.green = 0;
    background.blue = 0;
    int new_height;
    int new_width;

    int currentResolution = image->rows;  
    int currentCols = image->columns;

    //loop to do the upscaling until the current resolution equals or exceeds the target resolution
    while (currentResolution < target_resolution) {

        new_height = 2 * currentResolution;
        new_width = 2 * currentCols;

        //make a new image of the right size for the upscaled image
        ImageInfo* image_info;
        image_info=CloneImageInfo((ImageInfo *) NULL);
        upscaled_image = NewMagickImage(image_info, new_width, new_height, &background); 

        PixelPacket* originalPixels = GetImagePixels(new_image, 0, 0, new_width / 2, new_height / 2);
        PixelPacket* updatedPixels = SetImagePixels(upscaled_image, 0, 0, new_width, new_height);

        PixelPacket* originalPixelsGPU;
        PixelPacket* updatedPixelsGPU;

        //allocate space on the GPU for the not-fully-upscaled image
        if(hipMalloc(&originalPixelsGPU, sizeof(PixelPacket)*(new_width / 2 * new_height / 2))!= hipSuccess){
            fprintf(stderr, "Failed to allocate original image on GPU\n");
            exit(2);
        }
        //copy the not-fully-upscaled image to the GPU
        if(hipMemcpy(originalPixelsGPU, originalPixels, sizeof(PixelPacket)*(new_width / 2 * new_height / 2), hipMemcpyHostToDevice) != hipSuccess) {
            fprintf(stderr, "Failed to copy original image to the GPU\n");
            exit(2);
        }
        //allocate space on the GPU for the upscaled image
        if(hipMalloc(&updatedPixelsGPU, sizeof(PixelPacket)*(new_width * new_height)) != hipSuccess){
            fprintf(stderr, "Failed to allocate updated image on GPU\n");
            exit(2);
        }
        //copy the upscaled image to the GPU so that the fields are filled in
        if(hipMemcpy(updatedPixelsGPU, updatedPixels, sizeof(PixelPacket)*(new_width * new_height), hipMemcpyHostToDevice) != hipSuccess) {
            fprintf(stderr, "Failed to copy updated image to the GPU\n");
            exit(2);
        }
        
        //invoke the kernel
        gpu_upscale<<<new_width, new_height>>>(originalPixelsGPU, updatedPixelsGPU, new_width, new_height);  // THINK ABOUT THIS

        //allows all threads on the GPU to finish before continuing
        if (hipDeviceSynchronize() != hipSuccess) {
        fprintf(stderr, "Cuda Error: %s\n", hipGetErrorString(hipPeekAtLastError()));
        }

        //copy the upscaled image from GPU memory to the CPU memory
        if(hipMemcpy(updatedPixels, updatedPixelsGPU, sizeof(PixelPacket)*(new_width * new_height), hipMemcpyDeviceToHost) != hipSuccess) {
            fprintf(stderr, "Failed to copy updated image from the GPU\n");
            exit(2);
        }

        //Save image changes
        if(SyncAuthenticPixels(upscaled_image, exception) == MagickFalse){
            fprintf(stderr, "Failed to save image changes\n");
            exit(2);
        }
        //prepare for the next iteration of the While loop
        new_image = upscaled_image;
        currentResolution *= 2;
        currentCols *= 2;
    }
    return new_image;
}

/*
Procedure:           gpu_upscale
Parameters:          PixelPacket* originalPixels - the array of pixels of the non-upscaled image
                     PixelPacket* updatedPixelsGPU - the array of pixels of the image to be upscaled
                     int new_width - the horizontal resolution of the image whose pixels are being filled in
                     int new_height - the vertical resolution of the image whose pixels are being filled in
Purpose:             To change the colors of pixels in the output image in a manner consistent with the
                     bilinear interpolation algorithm for image upscaling
Produces:            Nothing (void) - the updatedPixelsGPU is modified to be an upscaled image
Preconditions:       originalPixels and updatedPixelsGPU must point to GPU memory.
                     new_width and new_height must be the correct resolutions for the updatedPixelsGPU array
                     (and thus, two times the resolution of the originalPixels array)
Postconditions:      updatedPixelsGPU is updated to be the upscaled image based on the bilinear interpolation
                     algorithm.
*/
__global__ void gpu_upscale(PixelPacket* originalPixels, PixelPacket* updatedPixelsGPU, int new_width, int new_height) {

    int row = threadIdx.x;
    int col = blockIdx.x;
    int scaledOldRow = 2 * row + 1;
    int scaledOldCol = 2 * col + 1;

    int oldRow = scaledOldRow / 4;
    int oldCol = scaledOldCol / 4;

    //if a thread is at an edge or corner where there don't exist enough pixels to calculate what the new
    //pixel's value is, then simply return.
    if(row==0 || row==new_height-1 || col==0 || col==new_width-1){
        return;
    }

    //below is the actual implementation of the bilinear interpolation algorithm.
    if(scaledOldRow % 4 == 1){       // upper
        if(scaledOldCol % 4 == 1){
            // upper left
            float upperRed = 0.75 * originalPixels[new_width / 2 * oldRow + oldCol].red + 0.25 * originalPixels[new_width / 2 * oldRow + (oldCol - 1)].red;
            float upperGreen = 0.75 * originalPixels[new_width / 2 * oldRow + oldCol].green + 0.25 * originalPixels[new_width / 2 * oldRow + (oldCol - 1)].green;
            float upperBlue = 0.75 * originalPixels[new_width / 2 * oldRow + oldCol].blue + 0.25 * originalPixels[new_width / 2 * oldRow + (oldCol - 1)].blue;

            float lowerRed = 0.75 * originalPixels[new_width / 2 * (oldRow - 1) + oldCol].red + 0.25 * originalPixels[new_width / 2 * (oldRow - 1) + (oldCol - 1)].red;
            float lowerGreen = 0.75 * originalPixels[new_width / 2 * (oldRow - 1) + oldCol].green + 0.25 * originalPixels[new_width / 2 * (oldRow - 1) + (oldCol - 1)].green;
            float lowerBlue = 0.75 * originalPixels[new_width / 2 * (oldRow - 1) + oldCol].blue + 0.25 * originalPixels[new_width / 2 * (oldRow - 1) + (oldCol - 1)].blue;

            int actualRed = (int) (0.75 * lowerRed + 0.25 * upperRed);
            int actualGreen = (int) (0.75 * lowerGreen + 0.25 * upperGreen);
            int actualBlue = (int) (0.75 * lowerBlue + 0.25 * upperBlue);

            updatedPixelsGPU[new_width * row + col].red = actualRed;
            updatedPixelsGPU[new_width * row + col].green = actualGreen;
            updatedPixelsGPU[new_width * row + col].blue = actualBlue;
        }else{
            // upper right
            float upperRed = 0.75 * originalPixels[new_width / 2 * oldRow + oldCol].red + 0.25 * originalPixels[new_width / 2 * oldRow + (oldCol + 1)].red;
            float upperGreen = 0.75 * originalPixels[new_width / 2 * oldRow + oldCol].green + 0.25 * originalPixels[new_width / 2 * oldRow + (oldCol + 1)].green;
            float upperBlue = 0.75 * originalPixels[new_width / 2 * oldRow + oldCol].blue + 0.25 * originalPixels[new_width / 2 * oldRow + (oldCol + 1)].blue;

            float lowerRed = 0.75 * originalPixels[new_width / 2 * (oldRow - 1) + oldCol].red + 0.25 * originalPixels[new_width / 2 * (oldRow - 1) + (oldCol + 1)].red;
            float lowerGreen = 0.75 * originalPixels[new_width / 2 * (oldRow - 1) + oldCol].green + 0.25 * originalPixels[new_width / 2 * (oldRow - 1) + (oldCol + 1)].green;
            float lowerBlue = 0.75 * originalPixels[new_width / 2 * (oldRow - 1) + oldCol].blue + 0.25 * originalPixels[new_width / 2 * (oldRow - 1) + (oldCol + 1)].blue;

            int actualRed = (int) (0.75 * lowerRed + 0.25 * upperRed);
            int actualGreen = (int) (0.75 * lowerGreen + 0.25 * upperGreen);
            int actualBlue = (int) (0.75 * lowerBlue + 0.25 * upperBlue);

            updatedPixelsGPU[new_width * row + col].red = actualRed;
            updatedPixelsGPU[new_width * row + col].green = actualGreen;
            updatedPixelsGPU[new_width * row + col].blue = actualBlue;
        }
    }else{                           // lower
        if(scaledOldCol % 4 == 1){
            // lower left
            float upperRed = 0.75 * originalPixels[new_width / 2 * oldRow + oldCol].red + 0.25 * originalPixels[new_width / 2 * oldRow + (oldCol - 1)].red;
            float upperGreen = 0.75 * originalPixels[new_width / 2 * oldRow + oldCol].green + 0.25 * originalPixels[new_width / 2 * oldRow + (oldCol - 1)].green;
            float upperBlue = 0.75 * originalPixels[new_width / 2 * oldRow + oldCol].blue + 0.25 * originalPixels[new_width / 2 * oldRow + (oldCol - 1)].blue;

            float lowerRed = 0.75 * originalPixels[new_width / 2 * (oldRow + 1) + oldCol].red + 0.25 * originalPixels[new_width / 2 * (oldRow + 1) + (oldCol - 1)].red;
            float lowerGreen = 0.75 * originalPixels[new_width / 2 * (oldRow + 1) + oldCol].green + 0.25 * originalPixels[new_width / 2 * (oldRow + 1) + (oldCol - 1)].green;
            float lowerBlue = 0.75 * originalPixels[new_width / 2 * (oldRow + 1) + oldCol].blue + 0.25 * originalPixels[new_width / 2 * (oldRow + 1) + (oldCol - 1)].blue;

            int actualRed = (int) (0.75 * lowerRed + 0.25 * upperRed);
            int actualGreen = (int) (0.75 * lowerGreen + 0.25 * upperGreen);
            int actualBlue = (int) (0.75 * lowerBlue + 0.25 * upperBlue);

            updatedPixelsGPU[new_width * row + col].red = actualRed;
            updatedPixelsGPU[new_width * row + col].green = actualGreen;
            updatedPixelsGPU[new_width * row + col].blue = actualBlue;
        }else{
            // lower right
           float upperRed = 0.75 * originalPixels[new_width / 2 * oldRow + oldCol].red + 0.25 * originalPixels[new_width / 2 * oldRow + (oldCol + 1)].red;
            float upperGreen = 0.75 * originalPixels[new_width / 2 * oldRow + oldCol].green + 0.25 * originalPixels[new_width / 2 * oldRow + (oldCol + 1)].green;
            float upperBlue = 0.75 * originalPixels[new_width / 2 * oldRow + oldCol].blue + 0.25 * originalPixels[new_width / 2 * oldRow + (oldCol + 1)].blue;

            float lowerRed = 0.75 * originalPixels[new_width / 2 * (oldRow + 1) + oldCol].red + 0.25 * originalPixels[new_width / 2 * (oldRow + 1) + (oldCol + 1)].red;
            float lowerGreen = 0.75 * originalPixels[new_width / 2 * (oldRow + 1) + oldCol].green + 0.25 * originalPixels[new_width / 2 * (oldRow + 1) + (oldCol + 1)].green;
            float lowerBlue = 0.75 * originalPixels[new_width / 2 * (oldRow + 1) + oldCol].blue + 0.25 * originalPixels[new_width / 2 * (oldRow + 1) + (oldCol + 1)].blue;

            int actualRed = (int) (0.75 * lowerRed + 0.25 * upperRed);
            int actualGreen = (int) (0.75 * lowerGreen + 0.25 * upperGreen);
            int actualBlue = (int) (0.75 * lowerBlue + 0.25 * upperBlue);

            updatedPixelsGPU[new_width * row + col].red = actualRed;
            updatedPixelsGPU[new_width * row + col].green = actualGreen;
            updatedPixelsGPU[new_width * row + col].blue = actualBlue;
        }
    }
}
